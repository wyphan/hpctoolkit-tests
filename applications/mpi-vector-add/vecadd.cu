
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define FAILURE -1
#define SUCCESS  0

#define TYPED_ALLOC(type, name, size) \
	type name = (type) malloc(size); \
	if (name == NULL) { \
          fprintf(stderr, "failed to allocate host vector " #name ".\n"); \
          exit(FAILURE); \
        }

#define CUDA_CALL(call, format) \
	{ hipError_t err = call; \
	  if (err != hipSuccess) { \
            fprintf(stderr, format " error: %s\n",  hipGetErrorString(err)); \
            exit(FAILURE); \
          } \
        }

__global__ void
vecadd_kernel(int *C, const int *A, const int *B, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}


int vecadd(int device) {

    if (device > 0) CUDA_CALL(hipSetDevice(device), "failed to set CUDA device");

    int n = 10000;
    size_t size = n * sizeof(int);

    printf("add two integer vectors of %d elements on a GPU\n", n);

    // allocate host vectors
    TYPED_ALLOC(int *, h_A, size);
    TYPED_ALLOC(int *, h_B, size);
    TYPED_ALLOC(int *, h_C, size);

    // initialize host input vectors h_A and h_B
    memset(h_A, 1, size); 
    memset(h_B, 2, size); 

    // allocate GPU vectors
    int *d_A = NULL;
    int *d_B = NULL;
    int *d_C = NULL;
    CUDA_CALL(hipMalloc((void **)&d_A, size), "failed to allocate GPU vector d_A.");
    CUDA_CALL(hipMalloc((void **)&d_B, size), "failed to allocate GPU vector d_B.");
    CUDA_CALL(hipMalloc((void **)&d_C, size), "failed to allocate GPU vector d_C.");

    printf("copy h_A and h_B from host to GPU\n");
    CUDA_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "failed to copy vector h_A from host to GPU.");
    CUDA_CALL(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "failed to copy vector h_B from host to GPU.");

    // launch the vector add kernel
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    printf("launch vecadd_kernel with %d blocks of %d threads\n", blocks, threads);
    vecadd_kernel<<<blocks, threads>>>(d_C, d_A, d_B, n);

    CUDA_CALL(hipGetLastError(), "failed to launch vecadd_kernel.");

    // copy the result from GPU to host 
    printf("copy d_C from GPU to the host\n");
    CUDA_CALL(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost), "failed to copy vector d_C from GPU to host.");

    // verify the result
    for (int i = 0; i < n; ++i) {
        if (h_A[i] + h_B[i] != h_C[i]) {
            fprintf(stderr, "bad sum: h_C[%d]=%d\n", i, h_C[i]);
            exit(FAILURE);
        }
    }

    // free vectors on host
    free(h_A);
    free(h_B);
    free(h_C);

    // free vectors on GPU
    CUDA_CALL(hipFree(d_A), "failed to free GPU vector d_A.");
    CUDA_CALL(hipFree(d_B), "failed to free GPU vector d_B.");
    CUDA_CALL(hipFree(d_C), "failed to free GPU vector d_C.");

    return 0;
}

